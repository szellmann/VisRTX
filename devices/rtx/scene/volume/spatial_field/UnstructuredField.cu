/*
 * Copyright (c) 2019-2023 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice,
 * this list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * 3. Neither the name of the copyright holder nor the names of its
 * contributors may be used to endorse or promote products derived from
 * this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
 * ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE
 * LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
 * CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
 * SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
 * INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
 * CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
 * ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
 * POSSIBILITY OF SUCH DAMAGE.
 */

#include "UnstructuredField.h"
// cuda
#include <hip/hip_math_constants.h>
// thrust
#include <thrust/for_each.h>
#include <thrust/tuple.h>
#include <thrust/iterator/counting_iterator.h>

namespace visrtx {

// Helper functions ///////////////////////////////////////////////////////////

// UnstructuredField definitions //////////////////////////////////////////////

UnstructuredField::UnstructuredField(DeviceGlobalState *d)
    : SpatialField(d)
{}

UnstructuredField::~UnstructuredField()
{
  cleanup();
}

void UnstructuredField::commit()
{
  cleanup();

  m_params.vertexPosition = getParamObject<Array1D>("vertex.position");
  m_params.vertexData = getParamObject<Array1D>("vertex.data");
  m_params.index = getParamObject<Array1D>("index");
  m_params.cellIndex = getParamObject<Array1D>("cell.index");

  if (!m_params.vertexPosition) {
    reportMessage(ANARI_SEVERITY_WARNING,
        "missing required parameter 'vertex.position' on unstructured spatial field");
    return;
  }

  if (!m_params.vertexData) { // currently vertex data only!
    reportMessage(ANARI_SEVERITY_WARNING,
        "missing required parameter 'vertex.data' on unstructured spatial field");
    return;
  }

  if (!m_params.index) {
    reportMessage(ANARI_SEVERITY_WARNING,
        "missing required parameter 'index' on unstructured spatial field");
    return;
  }

  if (!m_params.cellIndex) {
    reportMessage(ANARI_SEVERITY_WARNING,
        "missing required parameter 'cell.index' on unstructured spatial field");
    return;
  }

  // TODO: check data type/index type validity!
  // cf. stagingBuffer in SR field?

  // Calculate bounds //

  size_t numCells = m_params.cellIndex->size();
  m_aabbs.resize(numCells);

  auto begin = thrust::counting_iterator<uint64_t>(0);
  auto end = begin + numCells;

  auto *vertexPosition = m_params.vertexPosition->beginAs<vec3>(AddressSpace::GPU);
  auto *index = m_params.index->beginAs<uint64_t>(AddressSpace::GPU);
  auto *cellIndex = m_params.cellIndex->beginAs<uint64_t>(AddressSpace::GPU);

  size_t numIndices = m_params.index->endAs<uint64_t>(AddressSpace::GPU)-index;

  auto &state = *deviceState();

  thrust::transform(thrust::cuda::par.on(state.stream),
      begin,
      end,
      m_aabbs.begin(),
      [=] __device__(uint64_t cellID) {
        uint64_t firstIndex = cellIndex[cellID];
        uint64_t lastIndex = cellID < numCells-1 ? cellIndex[cellID+1] : numIndices;

        box3 result(vec3(HIP_INF_F), vec3(-HIP_INF_F));
        for (uint64_t i = firstIndex; i < lastIndex; ++i) {
          uint64_t idx = index[i];
          result.extend(vertexPosition[idx]);
        }
        return result;
      });

  m_aabbsBufferPtr = (hipDeviceptr_t)thrust::raw_pointer_cast(m_aabbs.data());

  std::vector<OptixBuildInput> obi(1);
  obi[0] = buildInput();

  reportMessage(ANARI_SEVERITY_DEBUG, "visrtx::UnstructuredField building cell BVH");
  buildOptixBVH(obi,
      m_bvhCells,
      m_traversableCells,
      m_cellBounds,
      this);

  buildGrid();

  upload();
}

box3 UnstructuredField::bounds() const
{
  return m_cellBounds;
}

float UnstructuredField::stepSize() const
{
  return 0.005f; // TODO!!
}

OptixBuildInput UnstructuredField::buildInput() const
{
  OptixBuildInput obi = {};

  obi.type = OPTIX_BUILD_INPUT_TYPE_CUSTOM_PRIMITIVES;

  obi.customPrimitiveArray.aabbBuffers = &m_aabbsBufferPtr;
  obi.customPrimitiveArray.numPrimitives = m_aabbs.size();

  static uint32_t buildInputFlags[1] = {OPTIX_GEOMETRY_FLAG_NONE};

  obi.customPrimitiveArray.flags = buildInputFlags;
  obi.customPrimitiveArray.numSbtRecords = 1;

  return obi;
}

bool UnstructuredField::isValid() const
{
  return true;
}

SpatialFieldGPUData UnstructuredField::gpuData() const
{
  SpatialFieldGPUData sf;
  sf.type = SpatialFieldType::UNSTRUCTURED;
  sf.data.unstructured.vertexData
      = m_params.vertexData->beginAs<float>(AddressSpace::GPU);
  sf.data.unstructured.cellsTraversable = m_traversableCells;
  sf.grid = m_uniformGrid.gpuData();
  return sf;
}

void UnstructuredField::cleanup()
{
  m_uniformGrid.cleanup();
}

void UnstructuredField::buildGrid()
{
}

} // namespace visrtx
